#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <Rinternals.h>
#include <iostream>

using namespace std;
// treat it as C code
extern "C" {
    SEXP gpu(SEXP ra, SEXP rk);
}

void mat(const double*A , const double* B, double* C, const int N, const int M, const int K) {
    int i,j,l;

    for(i=0; i<N; i++) {
        for(l=0; l<M; l++) {
            double a = A[M*i+l];
            for(j=0; j<K; j++) {
                C[K*i + j] += a*B[K*l+j];
            }
        }
    }
}

#define BLOCKSIZE 32

void nmfInit(double *a, int r, int c, int k, double *res);

SEXP gpu(SEXP ra, SEXP rk) {
	
    int k = INTEGER(rk)[0]; //k
    SEXP adim = 
       getAttrib(ra, R_DimSymbol);
    int m = INTEGER(adim)[0]; //how many rows
	int n = INTEGER(adim)[1];
	
    double *a = REAL(ra);
	SEXP rres = PROTECT(allocMatrix(REALSXP, m, n));
	double *res = REAL(rres);
	
	for (int i=0; i< m*n; i++)
		res[i] = 0.0;

	double *tmp = new double[m*n];
	double *tmp2 = new double[m*n];

	for (int i=0; i<m*n; i++)
		tmp2[i]= 0;
	
	for (int i=0; i<m; i++)
		for (int j=0; j<n; j++)
			tmp[i*n+j] = a[j*m+i];
	
	nmfInit(tmp, m, n, k, tmp2);

	for (int i=0; i<m; i++)
		for (int j=0; j<n; j++)
			res[j*m+i] = tmp2[i*n+j];
		
    return rres;
}

__global__ void nmfw(double *a, int r, int c, int k, double *w, double *h, double *wcp)//must be block synchronized!!!
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	//compute W
	if (col < k && row < r) {
		//ah'
		double sum = 0.0;
		double temp = 0.0;
		for (int i = 0; i < c; i++)
			sum += a[row*c + i]*h[col*c + i];
		temp = w[row*k+col]*sum;
		//whh'
		sum = 0.0;
		for (int i = 0; i < c; i++) {
			for (int j = 0; j < k; j++) {
				sum += w[row*k + j]*h[j*c + i]*h[col*c+i];
			}
		}
		__syncthreads();
		wcp[row*k+col] = temp/sum;
	}
}

__global__ void nmfh(double *a, int r, int c, int k, double *w, double *h, double *hcp)//must be block synchronized!!!
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	//compute H
	if (row < k && col < c) {
		//w'a
		double temp = 0.0;
		double sum;
		sum = 0.0;
		for (int i = 0; i < r; i++)
			sum += w[i*k + row]*a[i*c+col];

		temp = h[row*c+col]*sum;
		//w'wh
		sum = 0.0;
		for (int i = 0; i < k; i++)
			for (int j = 0; j < r; j++) 
				sum += w[j*k + row]*w[j*k + i]*h[i*c+col];

		__syncthreads();		
		hcp[row*c+col] = temp/sum;
	}
}

__global__ void nmfcpy(double *mat, double *matcp, int m, int n) //kernel copy must be block synchronized!!!
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (row < m && col < n)
		mat[row*n+col] = matcp[row*n+col];
}

void nmfInit(double *a, int r, int c, int k, double *tmp)
{
	const dim3 block(BLOCKSIZE, BLOCKSIZE);
	const dim3 grid((c + BLOCKSIZE - 1)/ BLOCKSIZE,(r + BLOCKSIZE - 1)/ BLOCKSIZE);
	const int niters = 100;
	srand(time(0));

	//initialize
	double *w = new double[r*k];
	double *h = new double[k*c];
	for (int i = 0; i < r*k; i++)
		w[i] = (double)rand()/RAND_MAX;
	for (int i = 0; i < k*c; i++)
		h[i] = (double)rand()/RAND_MAX;
	//initialize
	double *dev_w, *dev_h, *dev_a, *dev_wcp, *dev_hcp;

	hipMalloc((void**)&dev_w, sizeof(double)*r*k);
	hipMalloc((void**)&dev_h, sizeof(double)*k*c);
	hipMalloc((void**)&dev_wcp, sizeof(double)*r*k);
	hipMalloc((void**)&dev_hcp, sizeof(double)*k*c);
	hipMalloc((void**)&dev_a, sizeof(double)*r*c);
	hipMemcpy(dev_w, w, sizeof(double)*r*k, hipMemcpyHostToDevice);
	hipMemcpy(dev_wcp, w, sizeof(double)*r*k, hipMemcpyHostToDevice);
	hipMemcpy(dev_h, h, sizeof(double)*k*c, hipMemcpyHostToDevice);
	hipMemcpy(dev_hcp, h, sizeof(double)*k*c, hipMemcpyHostToDevice);
	hipMemcpy(dev_a, a, sizeof(double)*r*c, hipMemcpyHostToDevice);
	//
	//kernel
	for (int i=0; i<niters; i++) { //slow way
		nmfw<<<grid, block>>>(dev_a, r, c, k, dev_w, dev_h, dev_wcp);
		nmfcpy<<<grid, block>>>(dev_w, dev_wcp, r, k);
		nmfh<<<grid, block>>>(dev_a, r, c, k, dev_w, dev_h, dev_hcp);
		nmfcpy<<<grid, block>>>(dev_h, dev_hcp, k, c);
	}
	
	//cpy back
	hipMemcpy(w, dev_w, sizeof(double)*r*k, hipMemcpyDeviceToHost);
	hipMemcpy(h, dev_h, sizeof(double)*k*c, hipMemcpyDeviceToHost);

	//clean up
	hipFree(dev_w);
	hipFree(dev_h);
	hipFree(dev_a);

	mat(w, h, tmp, r, k, c);
}


