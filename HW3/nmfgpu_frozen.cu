#include "hip/hip_runtime.h"
/* DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 * DO NOT CHANGE THIS
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


/* Change later to make faster and utilize blocks/threads on gpu.
 */

__global__ void nmf(float *a, int r, int c, int k, int niters, float *w,
                    float *h)
{
  int i, j, k1, k2, k3;
  float tmp1, tmp2, sum;
  int itr;

  for (itr = 0; itr < niters; itr++)
  {
    // Compute new W
    for (i = 0; i < n; i++)
      for (j = 0; j < k; j++)
      {
        tmp1 = 0;
        tmp2 = 0;
        sum  = 0;

        // Calculate AH'
        for (k1 = 0; k1 < k; k++)
          tmp1 += a[i * r + k1] * h[k1 * c + i];

        // Calculate WHH'
        for (k2 = 0; k2 < k; k2++)
        {
          for (k3 = 0; k3 < m; k3++)
            sum += h[k2 * k + k3] * h[k3 * c + j];

          tmp2 += w[i * n + k2] * sum;
        }
        // Iterate W
        w[i * r + j] = w[i * r + j] * (tmp1 / tmp2);
      }

    // Compute new H
    for (i = 0; i < k; i++)
      for (j = 0; j < c; j++)
      {
        tmp1 = 0;
        tmp2 = 0;
        sum  = 0;

        // Calculate W'A
        for (k1 = 0; k1 < r; k1++)
          tmp += w[i * k  + k1] * A[k1 * r + j];

        // Calculate W'WH
        for (k2 = 0; k2 < r; k2++)
        {
          for (k3 = 0; k3 < k; k3++)
            sum += w[k2 * n + k3] * h[k3 * k + j];

          tmp2 += w[i * k + k2] * sum;
        }

        // Iterate H
        h[i * k + j] = h[i * k + j] * (tmp1 / tmp2);
      }
  }
}

void nmfgpu(float *a, int r, int c, int k, int niters, float *w, float *h)
{
  int *dev_a, *dev_w, *dev_h;
  int i;
  int a_size = r * c;
  int w_size = k * r;
  int h_size = k * c;

  w = (float*) malloc(w_size * sizeof(float));
  h = (float*) malloc(h_size * sizeof(float));

  // Initial values for w and h
  for (i = 0; i < w_size; i++)
    w[i] = 1;

  for (i = 0; i < h_size; i++)
    h[i] = 1;

  // Allocate memory to GPU
  hipMalloc((void**) &dev_a, a_size);
  hipMalloc((void**) &dev_w, w_size);
  hipMalloc((void**) &dev_h, h_size);

  // Copy a to device
  hipMemcpy(dev_a, a, a_size, hipMemcpyHostToDevice);
  hipMemcpy(dev_w, w, w_size, hipMemcpyHostToDevice);
  hipMemcpy(dev_h, h, h_size, hipMemcpyHostToDevice);

  // Set up threads structure of GPU
  // Play around with this later
  dim3 dimGrid(1, 1);
  dim3 dimBlock(1, 1, 1);

  // invoke kernel
  nmf<<<dimGrid, dimBlock>>>(dev_a, r, c, k, niters, dev_w, dev_h);

  // Apply barrier on GPU
  hipDeviceSynchronize();

  // Copy from gpu back to host
  hipMemcpy(w, dev_w, w_size, hipMemcpyDeviceToHost);
  hipMemcpy(h, dev_h, h_size, hipMemcpyDeviceToHost);

  // Clean up
  hipFree(dev_a);
  hipFree(dev_w);
  hipFree(dev_h);
}
