#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>

using namespace std;

#define BLOCKSIZE 16

/*void mat(const float*A , const float* B, float* C, const int N, const int M, const int K) {
      int i,j,l;
          #pragma omp parallel for shared(A,B,C) private(i,j,l)
              for(i=0; i<N; i++) {
                      for(l=0; l<M; l++) {
                                  float a  = A[M*i+l];
                                              for(j=0; j<K; j++) {
                                                              C[K*i + j] += a*B[K*l+j];
                                                                          }
                                                                                  }
                                                                                      }
                                                                                      }*/

__global__ void nmf(float *a, int r, int c, int k, int niters, float *w, float *h)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
      int col = blockIdx.x*blockDim.x + threadIdx.x;
        float temp = 0.0;
          float sum = 0.0;
            
            for (int iter = 0; iter < niters; iter++) {
                //compute W
                  if (col < k && row < r) {
                          //ah'
                          sum = 0.0;
                                for (int i = 0; i < c; i++)
                                          sum += a[row*c + i]*h[col*c + i];
                                      temp =  w[row*k+col]*sum;
                                            //whh'
                                            sum = 0.0;
                                                  for (int i = 0; i < c; i++) {
                                                            float sum2 = 0.0;
                                                                    for (int j = 0; j < k; j++) 
                                                                                sum2 += w[row*k + j]*h[j*c + i];
                                                                            sum += sum2*h[col*c+i];
                                                                                  }
                                                        __syncthreads();    
                                                              w[row*k+col] = temp/sum;
                                                                  }
                      __syncthreads();

                          //compute H
                          if (row < k && col < c) {
                                  //w'a
                                  temp = 0.0;
                                        sum = 0.0;
                                              for (int i = 0; i < r; i++)
                                                        sum += w[i*k + row]*a[i*c + col];
                                                    temp = h[row*c + col]*sum;
                                                          //w'wh
                                                          sum = 0;
                                                                for (int i = 0; i < k; i++) {
                                                                          float sum2 = 0.0;
                                                                                  for (int j = 0; j < r; j++) 
                                                                                              sum2 += w[j*k + row]*w[j*k + i];
                                                                                          sum += sum2*h[i*c+col];
                                                                                                }
                                                                      __syncthreads();    
                                                                            h[row*c+col] = temp/sum;
                                                                                }
                              __syncthreads();
                                }
}


void nmfgpu(float *a, int r, int c, int k, int niters, float *w, float *h)
{
    const dim3 block(BLOCKSIZE, BLOCKSIZE);
      const dim3 grid((r + BLOCKSIZE-1) / BLOCKSIZE, (c + BLOCKSIZE-1) / BLOCKSIZE);

        //initialize
        float *dev_w, *dev_h, *dev_a; 
          hipMalloc((void**)&dev_w, sizeof(float)*r*k);
            hipMalloc((void**)&dev_h, sizeof(float)*k*c);
              hipMalloc((void**)&dev_a, sizeof(float)*r*c);
                hipMemcpy(dev_w, w, sizeof(float)*r*k, hipMemcpyHostToDevice);
                  hipMemcpy(dev_h, h, sizeof(float)*k*c, hipMemcpyHostToDevice);
                    hipMemcpy(dev_a, a, sizeof(float)*r*c, hipMemcpyHostToDevice);
                      //
                      //kernel

                      nmf<<<grid, block>>>(dev_a, r, c, k, niters, dev_w, dev_h);
                        hipDeviceSynchronize();
                          //cpy back

                          hipMemcpy(w, dev_w, sizeof(float)*r*k, hipMemcpyDeviceToHost);
                            hipMemcpy(h, dev_h, sizeof(float)*k*c, hipMemcpyDeviceToHost);

                              //clean up
                              hipFree(dev_w);
                                hipFree(dev_h);
                                  hipFree(dev_a);
}

/*int main()
  {
    srand(1000);
      float *a, *w, *h;
        int r = 3;
          int k = 2;
            int c = 3;
              a = new float[r*c];
                w = new float[r*k];
                  h = new float[k*c];
                    int count = 1;
                      for (int i = 0; i < r*c; i++)
                        {
                            a[i] = count++;
                              }
                                float wh = 0.1;
                                  for (int i = 0; i < r*k; i++)
                                    {
                                        w[i] = wh;
                                            wh+=0.2;
                                              }
                                                wh = 0.1;
                                                  for (int i = 0; i < k*c; i++)
                                                    {
                                                        h[i] = wh;
                                                            wh+=0.2;
                                                              }
                                                                
                                                                nmfgpu(a, r, c, k, 100, w, h);
                                                                  
                                                                  float *res = new float[r*c];
                                                                    for (int i = 0; i<r*c; i++)
                                                                        res[i] = 0;
                                                                          mat(w,h,res,r,k,c);
                                                                            
                                                                            float error = 0;
                                                                              for (int i=0; i<r*c;i++)
                                                                                  error += abs(res[i]-a[i]);
                                                                                    cout << error << endl;
                                                                                      
                                                                                      for (int i=0; i< r; i++) {
                                                                                          for (int j=0; j< c; j++) {
                                                                                                cout << res[i*c+j] << " ";
                                                                                                    }
                                                                                                        cout << endl;
                                                                                                          }
                                                                                                          }*/
