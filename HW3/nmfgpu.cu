#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void nmf(float *a, int r, int c, int k, int niters, float *w,
                    float *h)
{
  int i;
  int r_num = blockIdx.x;
  int c_num = blockIdx.y;

  for (i = 0; i < niters; i++)
  {
  }
}


void nmfgpu(float *a, int r, int c, int k, int niters, float *w, float *h)
{
  int *dev_a, *dev_w, *dev_h;
  int i;
  int a_size = r * c;
  int w_size = k * r;
  int h_size = k * c;

  w = (float*) malloc(w_size * sizeof(float));
  h = (float*) malloc(h_size * sizeof(float));

  // Initial values for w and h
  for (i = 0; i < w_size; i++)
    w[i] = 1;

  for (i = 0; i < h_size; i++)
    h[i] = 1;

  // Allocate memory to GPU
  hipMalloc((void**) &dev_a, a_size);
  hipMalloc((void**) &dev_w, w_size);
  hipMalloc((void**) &dev_h, h_size);

  // Copy a to device
  hipMemcpy(dev_a, a, a_size, hipMemcpyHostToDevice);
  hipMemcpy(dev_w, w, w_size, hipMemcpyHostToDevice);
  hipMemcpy(dev_h, h, h_size, hipMemcpyHostToDevice);

  // Set up threads structure of GPU
  // Play around with this later
  dim3 dimGrid(r, c);
  dim3 dimBlock(1, 1, 1);

  // invoke kernel
  nmf<<<dimGrid, dimBlock>>>(dev_a, r, c, k, niters, dev_w, dev_h);

  // Apply barrier on GPU
  hipDeviceSynchronize();

  // Copy from gpu back to host
  hipMemcpy(w, dev_w, w_size, hipMemcpyDeviceToHost);
  hipMemcpy(h, dev_h, h_size, hipMemcpyDeviceToHost);

  // Clean up
  hipFree(dev_a);
  hipFree(dev_w);
  hipFree(dev_h);
}
