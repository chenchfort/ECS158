#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream>
using namespace std;

#define BLOCKSIZE 32

//test code
/*void mat(const float*A , const float* B, float* C, const int N, const int M, const int K) {
    int i,j,l;
    #pragma omp parallel for shared(A,B,C) private(i,j,l)
    for(i=0; i<N; i++) {
        for(l=0; l<M; l++) {
            float a  = A[M*i+l];
            for(j=0; j<K; j++) {
                C[K*i + j] += a*B[K*l+j];
            }
        }
    }
}*/

__global__ void nmfw(float *a, int r, int c, int k, float *w, float *h, float *wcp)//must be block synchronized!!!
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	//compute W
	if (col < k && row < r) {
		//ah'
		float sum = 0.0;
		float temp = 0.0;
		for (int i = 0; i < c; i++)
			sum += a[row*c + i]*h[col*c + i];
		temp =  w[row*k+col]*sum;
		//whh'
		sum = 0.0;
		for (int i = 0; i < c; i++) {
			for (int j = 0; j < k; j++) {
				sum += w[row*k + j]*h[j*c + i]*h[col*c+i];
			}
		}
		__syncthreads();
		wcp[row*k+col] = temp/sum;
	}
}

__global__ void nmfh(float *a, int r, int c, int k, float *w, float *h, float *hcp)//must be block synchronized!!!
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	//compute H
	if (row < k && col < c) {
		//w'a
		float temp = 0.0;
		float sum;
		sum = 0.0;
		for (int i = 0; i < r; i++)
			sum += w[i*k + row]*a[i*c+col];

		temp = h[row*c+col]*sum;
		//w'wh
		sum = 0.0;
		for (int i = 0; i < k; i++)
			for (int j = 0; j < r; j++) 
				sum += w[j*k + row]*w[j*k + i]*h[i*c+col];

		__syncthreads();		
		hcp[row*c+col] = temp/sum;
	}
}

__global__ void nmfcpy(float *mat, float *matcp, int m, int n) //kernel copy must be block synchronized!!!
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (row < m && col < n)
		mat[row*n+col] = matcp[row*n+col];
}

void nmfgpu(float *a, int r, int c, int k, int niters, float *w, float *h)
{
	const dim3 block(BLOCKSIZE, BLOCKSIZE);
	const dim3 grid((c + BLOCKSIZE - 1)/ BLOCKSIZE,(r + BLOCKSIZE - 1)/ BLOCKSIZE);
	//initialize
	float *dev_w, *dev_h, *dev_a, *dev_wcp, *dev_hcp; 
	hipMalloc((void**)&dev_w, sizeof(float)*r*k);
	hipMalloc((void**)&dev_h, sizeof(float)*k*c);
	hipMalloc((void**)&dev_wcp, sizeof(float)*r*k);
	hipMalloc((void**)&dev_hcp, sizeof(float)*k*c);
	hipMalloc((void**)&dev_a, sizeof(float)*r*c);
	hipMemcpy(dev_w, w, sizeof(float)*r*k, hipMemcpyHostToDevice);
	hipMemcpy(dev_h, h, sizeof(float)*k*c, hipMemcpyHostToDevice);
	hipMemcpy(dev_a, a, sizeof(float)*r*c, hipMemcpyHostToDevice);
	//
	//kernel
	for (int i=0; i<niters; i++) { //slow way
		nmfw<<<grid, block>>>(dev_a, r, c, k, dev_w, dev_h, dev_wcp);
		hipDeviceSynchronize();
		nmfcpy<<<grid, block>>>(dev_w, dev_wcp, r, k);
		hipDeviceSynchronize();
		nmfh<<<grid, block>>>(dev_a, r, c, k, dev_w, dev_h, dev_hcp);
		hipDeviceSynchronize();
		nmfcpy<<<grid, block>>>(dev_h, dev_hcp, k, c);
		hipDeviceSynchronize();
	}
	
	//cpy back
	hipMemcpy(w, dev_w, sizeof(float)*r*k, hipMemcpyDeviceToHost);
	hipMemcpy(h, dev_h, sizeof(float)*k*c, hipMemcpyDeviceToHost);

	//clean up
	hipFree(dev_w);
	hipFree(dev_h);
	hipFree(dev_a);
}

//test code, u can test it if u want
/*int main()
{
	srand(1000);
	float *w, *h;
	const int r = 194;
	int k = 50;

	const int c = 259;

	w = new float[r*k];
	h = new float[k*c];

	float a[r*c];
	ifstream file("af.txt");
	for (int i = 0; i < 194 * 259; i++)
		file >> a[i];
	
	for (int i = 0; i < r*k; i++)
	{
		w[i] = (float)rand()/RAND_MAX;
	}
	for (int i = 0; i < k*c; i++)
	{
		h[i] = (float)rand()/RAND_MAX;
	}

	nmfgpu(a, r, c, k, 100, w, h);
	
	float *res = new float[r*c];

	for (int i = 0; i<r*c; i++)
		res[i] = 0;

	mat(w,h,res,r,k,c);
	
	ofstream output("result.txt");

	for (int i=0; i < r; i++) {
		for (int j=0; j <c; j++)
			output << res[i*c+j] << " ";
		output << "\n";
	}
	
	
}*/
