#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 32

__global__ void quad(float *a, int n, float *u, float *out)
{
  int tot_th = gridDim.x * blockDim.x; //Total number of threads
  int t_id   = blockIdx.x * blockDim.x + threadIdx.x; //Thread number
  int i, j;
  float sum = 0;

  #ifdef DEBUG
  printf("Thread : %d out of %d\n", t_id, tot_th);
  #endif

  // Perform matrix quad
  for (i = t_id; i < n; i += tot_th)
    for (j = 0; j < n; j++)
      sum += u[i] * a[i * n + j] * u[j];

  atomicAdd(out, sum);
}

float gpuquad(float *a, int n, float *u) {
    float *da, *du, *dout;
    float hout = 0;

    hipMalloc((void **)&da, n * n * sizeof(float));
    hipMalloc((void **)&du, n * sizeof(float));
    hipMalloc((void **)&dout, sizeof(float));

    hipMemcpy(da, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(du, u, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dout, &hout, sizeof(float), hipMemcpyHostToDevice);

    dim3 dimGrid(1, 1);
    dim3 dimBlock(n, 1, 1);

    quad<<<dimGrid, dimBlock>>>(da, n, du, dout);

    hipDeviceSynchronize();

    hipMemcpy(&hout, dout, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(du);

    return hout;
}

int main(void)
{
  int n = 2;
  int i, j;
  float *a = (float*) malloc(n * n * sizeof(float));
  float *u = (float*) malloc(n * sizeof(float));

  a[0] = 1;
  a[1] = 2;
  a[2] = 2;
  a[3] = 4;

  u[0] = 1;
  u[1] = 2;

  #ifdef DEBUG
  // Serial code for testing
  float sum = 0;
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      sum += u[i] * a[i * n + j] * u[j];
  printf("Solution = %f\n", sum);
  #endif

  float output = gpuquad(a, n, u);

  #ifdef DEBUG
  printf("GPU = %f\n", output);
  #endif

  free(a);
  free(u);

  return 0;
}

