#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;

#define BLOCKSIZE 32

__global__ void quad(float *a, int n, float *u) {

}

float gpuquad(float *a, int n, float *u) {
    int *da, *du;
    hipMalloc((void **)&da, n * n);
    hipMalloc((void **)&du, n);
    hipMemcpy(da, a, n * n, hipMemcpyHostToDevice);
    hipMemcpy(du, u, n, hipMemcpyHostToDevice);
    dim3 dimGrid(n, 1);
    dim3 dimBlock(1, 1, 1);
    quad<<<dimGrid,dimBlock>>>(da, n, du);
    hipDeviceSynchronize();

    hipFree(da);
    hipFree(du);
}

int main ()
{

}

