#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG
#define BLOCK 16

__global__ void quad(float *a, int n, float *u, float *v)
{
  int col  = blockIdx.x * blockDim.x + threadIdx.x; // x thread number
  int row  = blockIdx.y * blockDim.y + threadIdx.y; // y threaqd number

  if (row < n && col < n && col >= row) {
	float sum = u[col]*a[row*n+col]*u[row];
	if (col == row)
		atomicAdd(v, sum);
	else
		atomicAdd(v, 2*sum);
  }
}

float gpuquad(float *a, int n, float *u) {
  // Function to perform v = u'Au
    float *da, *du, *dv;
    float v = 0;

    hipMalloc((void **)&da, n * n * sizeof(float));
    hipMalloc((void **)&du, n * sizeof(float));
    hipMalloc((void **)&dv, sizeof(float));

    hipMemcpy(da, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(du, u, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dv, &v, sizeof(float), hipMemcpyHostToDevice);

    int size = (n+BLOCK-1) / BLOCK;

    dim3 dimGrid(size, size);     // Fine tune parameters later
    dim3 dimBlock(BLOCK, BLOCK);

    quad<<<dimGrid, dimBlock>>>(da, n, du, dv);
    hipMemcpy(&v, dv, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(du);
    hipFree(dv);

    return v;
}

int main(void)
{
  int n = 2;
  float *a = (float*) malloc(n * n * sizeof(float));
  float *u = (float*) malloc(n * sizeof(float));

  a[0] = 1;
  a[1] = 2;
  a[2] = 2;
  a[3] = 4;

  u[0] = 1;
  u[1] = 2;

  #ifdef DEBUG
  // Serial code for testing
  // Possiably true in general
  // Check input with R
  int i, j;
  float sum = 0;
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      sum += u[i] * a[i * n + j] * u[j];
  printf("Solution = %f\n", sum);
  #endif

  float output = gpuquad(a, n, u);

  #ifdef DEBUG
  printf("GPU = %f\n", output);
  #endif

  free(a);
  free(u);

  return 0;
}
