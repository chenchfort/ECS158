#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK 16

__global__ void quad(float *a, int n, float *u, float *v)
{
  int col  = blockIdx.x * blockDim.x + threadIdx.x;
  int row  = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < n && col < n && col >= row) {
	float sum = u[col]*a[row*n+col]*u[row];
	if (col == row)
		atomicAdd(v, sum);
	else
		atomicAdd(v, 2*sum);
  }
}

float gpuquad(float *a, int n, float *u) {
    float *da, *du, *dv;
    float v = 0;

    hipMalloc((void **)&da, n * n * sizeof(float));
    hipMalloc((void **)&du, n * sizeof(float));
    hipMalloc((void **)&dv, sizeof(float));

    hipMemcpy(da, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(du, u, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dv, &v, sizeof(float), hipMemcpyHostToDevice);

    int size = (n+BLOCK-1) / BLOCK;

    dim3 dimGrid(size, size);
    dim3 dimBlock(BLOCK, BLOCK);

    quad<<<dimGrid, dimBlock>>>(da, n, du, dv);
    hipMemcpy(&v, dv, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(du);
    hipFree(dv);

    return v;
}
