#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG
#define BLOCK 32

__global__ void quad(float *a, int n, float *u, float *v)
{
  int tot_x = gridDim.x * blockDim.x; // Total number of x threads
  int tot_y = gridDim.y * blockDim.y; // Total number of y threads
  int me_x  = blockIdx.x * blockDim.x + threadIdx.x; // x thread number
  int me_y  = blockIdx.y * blockDim.y + threadIdx.y; // y threaqd number

  int i, j;
  float sum = 0;

  // Perform matrix quad v = u'Au
  for (i = me_x; i < n; i += tot_x)
    for (j = me_y; j < n; j += tot_y)
      sum += u[i] * a[i * n + j] * u[j];

  // Add atomically to output
  atomicAdd(v, sum);
}

float gpuquad(float *a, int n, float *u) {
  // Function to perform v = u'Au
    float *da, *du, *dv;
    float v = 0;

    hipMalloc((void **)&da, n * n * sizeof(float));
    hipMalloc((void **)&du, n * sizeof(float));
    hipMalloc((void **)&dv, sizeof(float));

    hipMemcpy(da, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(du, u, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dv, &v, sizeof(float), hipMemcpyHostToDevice);

    int size = (n / BLOCK) + 1;

    dim3 dimGrid(size, size);     // Fine tune parameters later
    dim3 dimBlock(BLOCK, BLOCK, 1);

    quad<<<dimGrid, dimBlock>>>(da, n, du, dv);

    hipDeviceSynchronize();

    hipMemcpy(&v, dv, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(du);
    hipFree(dv);

    return v;
}

int main(void)
{
  int n = 2;
  float *a = (float*) malloc(n * n * sizeof(float));
  float *u = (float*) malloc(n * sizeof(float));

  a[0] = 1;
  a[1] = 2;
  a[2] = 2;
  a[3] = 4;

  u[0] = 1;
  u[1] = 2;

  #ifdef DEBUG
  // Serial code for testing
  // Possiably true in general
  // Check input with R
  int i, j;
  float sum = 0;
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      sum += u[i] * a[i * n + j] * u[j];
  printf("Solution = %f\n", sum);
  #endif

  float output = gpuquad(a, n, u);

  #ifdef DEBUG
  printf("GPU = %f\n", output);
  #endif

  free(a);
  free(u);

  return 0;
}

